#include "hip/hip_runtime.h"
__global__ void MatMul_TILE_BLOCK_GENERAL(float *input0, float *input1, float *input2, float *input3, float *output0){
	float *g_vec = input0;
	float *g_mat_data = input1;
	int *g_mat_index = (int*)input2;
	float *bias = input3;
	float *g_data = output0;
	const float SPARSITY = SPARSITY_VALUE;
    const int M = M_GLOBAL_VALUE;
    const int K = K_GLOBAL_VALUE;
    const int N = N_GLOBAL_VALUE;
    const int K_sparse = int(K * (1-SPARSITY));

    const int BLOCK_SIZE_M = BLOCK_SIZE_M_VALUE;
    const int BLOCK_SIZE_N = BLOCK_SIZE_N_VALUE;
    // BLOCK_SIZE_K should > NUM_BANK
    const int BLOCK_SIZE_K = BLOCK_SIZE_K_VALUE;
    const int THREAD_SIZE_M = THREAD_SIZE_M_VALUE;
    const int THREAD_SIZE_N = THREAD_SIZE_N_VALUE;

    const int ALIGN_N = BLOCK_SIZE_N;

    const int BANK_VAL = BANK_VAL_VALUE;
    const int NUM_BANK = K / BANK_VAL;

    const int BANK_NUM_PER_BLOCK = BLOCK_SIZE_K / BANK_VAL;
    const int BLOCK_SIZE_K_SPARSE = int(BLOCK_SIZE_K * (1-SPARSITY));
    const int LEN_OF_BANK_PER_SPARSE_BLOCK = BLOCK_SIZE_K_SPARSE / BANK_NUM_PER_BLOCK;

	int M_BLOCK_START = blockIdx.x * BLOCK_SIZE_M;
	int N_BLOCK_START = blockIdx.y * BLOCK_SIZE_N;

	

	const int A_THREADS_PER_ROW = BLOCK_SIZE_M / 4;
	const int B_THREADS_PER_ROW = BLOCK_SIZE_N / 4;

	const int THREADS_PER_BLOCK = (BLOCK_SIZE_M / THREAD_SIZE_M) * (BLOCK_SIZE_N / THREAD_SIZE_N);

	const int A_STRIDES = THREADS_PER_BLOCK / A_THREADS_PER_ROW;
	const int B_STRIDES = THREADS_PER_BLOCK / B_THREADS_PER_ROW;

	__shared__ float A_shared[BLOCK_SIZE_M * BLOCK_SIZE_K_SPARSE];
	__shared__ float B_shared[BLOCK_SIZE_N * BLOCK_SIZE_K_SPARSE];

	float A_reg[THREAD_SIZE_M];
	float B_reg[THREAD_SIZE_N];
	float C_reg[THREAD_SIZE_N][THREAD_SIZE_M] = {0};

	int tid = threadIdx.x;

	int t_N = tid % (BLOCK_SIZE_N / THREAD_SIZE_N);
	int t_M = tid / (BLOCK_SIZE_N / THREAD_SIZE_N);

	int A_BLOCK_ROW_START = tid / A_THREADS_PER_ROW;
	int B_BLOCK_ROW_START = tid / B_THREADS_PER_ROW;

	int A_BLOCK_COL_START = tid % A_THREADS_PER_ROW * 4;
	int B_BLOCK_COL_START = tid % B_THREADS_PER_ROW * 4;

	for(int K_BLOCK_START = 0, K_SPARSE_BLOCK_START = 0; K_BLOCK_START < K; K_BLOCK_START += BLOCK_SIZE_K, K_SPARSE_BLOCK_START += BLOCK_SIZE_K_SPARSE){
		float *A_global_ptr = g_vec + M_BLOCK_START;
		float *B_global_ptr = g_mat_data + K_SPARSE_BLOCK_START * N + N_BLOCK_START;
		int *B_index_global_ptr = g_mat_index + K_SPARSE_BLOCK_START * N + N_BLOCK_START;

		__syncthreads();

		#pragma unroll
		for(int i = 0; i < BLOCK_SIZE_K_SPARSE; i += A_STRIDES){
			int idx = *(B_index_global_ptr + (i + A_BLOCK_ROW_START) * N);
			*(float4 *)(A_shared + (i + A_BLOCK_ROW_START) * BLOCK_SIZE_M + A_BLOCK_COL_START) = 
				*(float4 *)(A_global_ptr + idx * M + A_BLOCK_COL_START);
		}

		#pragma unroll
		for(int i = 0; i < BLOCK_SIZE_K_SPARSE; i += B_STRIDES){
			*(float4 *)(B_shared + (i + B_BLOCK_ROW_START) * BLOCK_SIZE_N + B_BLOCK_COL_START) =
				*(float4 *)(B_global_ptr + (i + B_BLOCK_ROW_START) * N + B_BLOCK_COL_START);
		}

		__syncthreads();

		#pragma unroll
		for(int i = 0; i < BLOCK_SIZE_K_SPARSE; i += 1){
			#pragma unroll
			for(int k = 0; k < THREAD_SIZE_M; k += 1){
				A_reg[k] = A_shared[i * BLOCK_SIZE_M + t_M * THREAD_SIZE_M + k];
			}
			#pragma unroll
			for(int k = 0; k < THREAD_SIZE_N; k += 1){
				B_reg[k] = B_shared[i * BLOCK_SIZE_N + t_N * THREAD_SIZE_N + k];
			}
			#pragma unroll
			for(int k = 0; k < THREAD_SIZE_N; k += 1){
				#pragma unroll
				for(int j = 0; j < THREAD_SIZE_M; j += 1){
					C_reg[k][j] += B_reg[k] * A_reg[j];
				}
			}
		}
	}
	
	#pragma unroll
	for(int i = 0; i < THREAD_SIZE_N; i += 1){
		#pragma unroll
		for(int j = 0; j < THREAD_SIZE_M; j += 1){
			g_data[(BLOCK_SIZE_N * blockIdx.y + THREAD_SIZE_N * t_N + i) * M + (BLOCK_SIZE_M * blockIdx.x + THREAD_SIZE_M * t_M + j)] =
				C_reg[i][j];
		}
	}
}